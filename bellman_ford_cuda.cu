#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <limits.h>
#include <assert.h>
#include <sys/time.h>

#define INF 1000000

// #define CHECK(call)                                                           \
//     {                                                                         \
//         const hipError_t error = call;                                       \
//         if (error != hipSuccess)                                             \
//         {                                                                     \
//             fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);            \
//             fprintf(stderr, "code: %d, reason: %s\n", error,                  \
//                     hipGetErrorString(error));                               \
//             exit(1);                                                          \
//         }                                                                     \
//     }

#define VERTICES 983
int mat[VERTICES * VERTICES]; // the adjacency matrix

void abort_with_error_message(const char *msg) {
    fprintf(stderr, "%s\n", msg);
    exit(1);
}

int read_file(const char *filename) {
    char line[256];

    // Initial the matrix with INFINITY 
    for (int i = 0; i < VERTICES; i++){
        for (int j = 0; j < VERTICES; j++){
            if (i != j){
                mat[i * VERTICES + j] = INF; 
            }else{
                mat[i * VERTICES + j] = 0;
            }
        }
    }

    // Open the CSV file
    FILE* file = fopen(filename, "r");

    // Read each line in the CSV file and update the matrix
    while (fgets(line, sizeof(line), file)) {
        char* token;
        char* rest = line;
        int src_id, dest_id;
        float distance;

        // Tokenize the line based on the comma delimiter
        token = strtok_r(rest, ",", &rest);
        src_id = atoi(token);
        
        token = strtok_r(rest, ",", &rest);
        dest_id = atoi(token);
        
        token = strtok_r(rest, ",", &rest);
        distance = atoi(token);
        
        // Update the matrix with the distance value
        if (src_id < VERTICES && dest_id < VERTICES) {
            mat[src_id * VERTICES + dest_id] = distance;
        }  
        // printf("element: %d\n", mat[src_id * VERTICES + dest_id]);
    }
    return 0;
}

void print_result(bool has_negative_cycle, int *dist) {
    FILE *outputf = fopen("cuda_output.txt", "w");
    if (!has_negative_cycle) {
        for (int i = 0; i < VERTICES; i++) {
            if (dist[i] > INF)
                dist[i] = INF;
            fprintf("%d\n", dist[i]);
        }
        fflush(outputf);
    } else {
        printf("FOUND NEGATIVE CYCLE!\n");
    }
    fclose(outputf);
}

__global__ void bellman_ford_one_iter(int n, int *d_mat, int *d_dist, bool *d_has_next, int iter_num) {
    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    if (global_tid >= n)
        return;
    for (int u = 0; u < n; u++) {
        for (int v = global_tid; v < n; v += elementSkip) {
            int weight = d_mat[u * n + v];
            if (weight < INF) {
                int new_dist = d_dist[u] + weight;
                if (new_dist < d_dist[v]) {
                    *d_has_next = true;
                    d_dist[v] = new_dist;
                }
            }
        }
    }
}

void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);

    int iter_num = 0;
    int *d_mat, *d_dist;
    bool *d_has_next, h_has_next;

    hipMalloc(&d_mat, sizeof(int) * n * n);
    hipMalloc(&d_dist, sizeof(int) * n);
    hipMalloc(&d_has_next, sizeof(bool));

    *has_negative_cycle = false;

    for (int i = 0; i < n; i++) {
        dist[i] = INF;
    }

    dist[0] = 0;
    hipMemcpy(d_mat, mat, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_dist, dist, sizeof(int) * n, hipMemcpyHostToDevice);

    for (;;) {
        h_has_next = false;
        hipMemcpy(d_has_next, &h_has_next, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_one_iter<<<blocks, threads>>>(n, d_mat, d_dist, d_has_next, iter_num);
        hipDeviceSynchronize();
        hipMemcpy(&h_has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if (iter_num >= n - 1) {
            *has_negative_cycle = true;
            break;
        }
        if (!h_has_next) {
            break;
        }
    }
    if (!*has_negative_cycle) {
        hipMemcpy(dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);
    }

    hipFree(d_mat);
    hipFree(d_dist);
    hipFree(d_has_next);
}

int main(int argc, char **argv) {
    if (argc <= 1) {
        abort_with_error_message("INPUT FILE WAS NOT FOUND!");
    }
    if (argc <= 3) {
        abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
    }
    const char *filename = argv[1];
    int blockPerGrid = atoi(argv[2]);
    int threadsPerBlock = atoi(argv[3]);

    int dist[VERTICES];
    bool has_negative_cycle = false;
    
    read_file(filename);
    memset(dist, 0, sizeof(dist));

    // time counter
    timeval start_wall_time_t, end_wall_time_t;
    float ms_wall;
    hipDeviceReset();
    // start timer
    gettimeofday(&start_wall_time_t, NULL);
    // bellman-ford algorithm
    bellman_ford(blockPerGrid, threadsPerBlock, VERTICES, mat, dist, &has_negative_cycle);
    hipDeviceSynchronize();
    // end timer
    gettimeofday(&end_wall_time_t, NULL);
    ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000 +
               end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

    
    printf("Network Specifications----------\n");
    printf("Number of nodes:\t%d\n", VERTICES);
    printf("Number of edges:\t%d\n", n_edges);
    printf("OpenMP Specifications-----------\n");
    printf('Number of THREADS:\t%d\n', NUM_THREADS);
    printf("Exe time:\t%.6f sec\n", (ms_wall / 1000.0));
    printf("--------------------------------\n");
    print_result(has_negative_cycle, dist);

    return 0;
}
