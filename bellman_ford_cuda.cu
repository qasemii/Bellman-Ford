
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <limits.h>
#include <assert.h>
#include <time.h>
// #include "hpc.h"

#define INF 999999
#define VERTICES 10000 //total vertices 264,346


double gettime(void){
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

void read_file(const char* filename, int* weights, int* n_edges) {
    // Initialize the matrix with INF and 0 for diagonals
    for (int i = 0; i < VERTICES; i++) {
        for (int j = 0; j < VERTICES; j++) {
            if (i != j) {
                weights[i * VERTICES + j] = INF;
            } else {
                weights[i * VERTICES + j] = 0;
            }
        }
    }

    // Open the CSV file
    FILE* file = fopen(filename, "r");

    // Read each line in the CSV file and update the matrix
    char line[256];
    n_edges = 0;
    while (fgets(line, sizeof(line), file)) {
        char* token;
        char* rest = line;
        int src_id, dest_id, distance;

        // Tokenize the line based on the comma delimiter
        token = strtok_r(rest, ",", &rest);
        src_id = atoi(token);

        token = strtok_r(rest, ",", &rest);
        dest_id = atoi(token);

        token = strtok_r(rest, ",", &rest);
        distance = atoi(token);

        // Update the matrix with the distance value
        if (src_id < VERTICES && dest_id < VERTICES) {
            n_edges++;
            weights[src_id * VERTICES + dest_id] = distance;
        }
    }
    fclose(file);
}

void save_results(int *distance, bool has_negative_cycle) {
    FILE *outputf = fopen("cuda_output.txt", "w");
    if (!has_negative_cycle) {
        for (int i = 0; i < VERTICES; i++) {
            if (distance[i] > INT_MAX)
                distance[i] = INT_MAX;
            fprintf(outputf, "%d\n", distance[i]);
        }
        fflush(outputf);
    } else {
        fprintf(outputf, "Negative cycle detected!\n");
    }
    fclose(outputf);
}

// sequential ================================================================================================
__global__ void bellman_ford_sequential_kernel(int *d_weights, int *d_distance, bool *d_changed) {
    
    for (int u = 0; u < VERTICES; u++) {
        for (int v = 0; v < VERTICES; v++) {
            int weight = d_weights[u * VERTICES + v];
            if (weight < INF) {
                int new_distance = d_distance[u] + weight;
                if (new_distance < d_distance[v]) {
                    *d_changed = true;
                    d_distance[v] = new_distance;
                }
            }
        }
    }
}

void bellman_ford_sequential(int *weights, int *distance, int start, bool *has_negative_cycle) {

    int iter_num = 0;
    int *d_weights, *d_distance;
    bool *d_changed, h_changed;

    // initializing the distance array
    for (int i = 0; i < VERTICES; i++) {
        distance[i] = INF;
    }
    distance[start] = 0;

    // Allocate GPU memory for d_weights, d_distance, d_changed
    hipMalloc(&d_weights, sizeof(int) * VERTICES * VERTICES);
    hipMalloc(&d_distance, sizeof(int) * VERTICES);
    hipMalloc(&d_changed, sizeof(bool));

    //Transfer the data from host to GPU.
    hipMemcpy(d_weights, weights, sizeof(int) * VERTICES * VERTICES, hipMemcpyHostToDevice);
    hipMemcpy(d_distance, distance, sizeof(int) * VERTICES, hipMemcpyHostToDevice);

    for (;;) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_sequential_kernel<<<1, 1>>>(d_weights, d_distance, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if (iter_num >= VERTICES - 1) {
            *has_negative_cycle = true;
            break;
        }
        if (!h_changed) {
            break;
        }
    }
    if (!*has_negative_cycle) {
        // Copy the shortest path distances back to the host memory
        hipMemcpy(distance, d_distance, sizeof(int) * VERTICES, hipMemcpyDeviceToHost);
    }
    
    // Free up the GPU memory.
    hipFree(d_weights);
    hipFree(d_distance);
    hipFree(d_changed);
}

// bellman_ford_withBlocks ===================================================================================
__global__ void bellman_ford_withBlock_kernel(int *d_weights, int *d_distance, bool *d_changed) {
    int global_tid = blockIdx.x;

    if (global_tid < VERTICES){
        for (int u = 0; u < VERTICES; u++) {
            for (int v = global_tid; v < VERTICES; v += gridDim.x) {
                int weight = d_weights[u * VERTICES + v];
                if (weight < INF) {
                    int new_distance = d_distance[u] + weight;
                    if (new_distance < d_distance[v]) {
                        *d_changed = true;
                        d_distance[v] = new_distance;
                    }
                }
            }
        }
    }
}

void bellman_ford_withBlock(int *weights, int *distance, int start, bool *has_negative_cycle) {
    int iter_num = 0;
    int *d_weights, *d_distance;
    bool *d_changed, h_changed;

    // initializing the distance array
    for (int i = 0; i < VERTICES; i++) {
        distance[i] = INF;
    }
    distance[start] = 0;

    // Allocate GPU memory for d_weights, d_distance, d_changed
    hipMalloc(&d_weights, sizeof(int) * VERTICES * VERTICES);
    hipMalloc(&d_distance, sizeof(int) * VERTICES);
    hipMalloc(&d_changed, sizeof(bool));

    //Transfer the data from host to GPU.
    hipMemcpy(d_weights, weights, sizeof(int) * VERTICES * VERTICES, hipMemcpyHostToDevice);
    hipMemcpy(d_distance, distance, sizeof(int) * VERTICES, hipMemcpyHostToDevice);

    for (;;) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_withBlock_kernel<<<VERTICES, 1>>>(d_weights, d_distance, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if (iter_num >= VERTICES - 1) {
            *has_negative_cycle = true;
            break;
        }
        if (!h_changed) {
            break;
        }
    }
    if (!*has_negative_cycle) {
        // Copy the shortest path distances back to the host memory
        hipMemcpy(distance, d_distance, sizeof(int) * VERTICES, hipMemcpyDeviceToHost);
    }
    
    // Free up the GPU memory.
    hipFree(d_weights);
    hipFree(d_distance);
    hipFree(d_changed);
}

// bellman_ford_withThreads ==================================================================================
__global__ void bellman_ford_withThread_kernel(int *d_weights, int *d_distance, bool *d_changed) {

    for (int u = 0; u < VERTICES; u++) {
        for (int v = threadIdx.x; v < VERTICES; v += blockDim.x) {
            int weight = d_weights[u * VERTICES + v];
            if (weight < INF) {
                int new_distance = d_distance[u] + weight;
                if (new_distance < d_distance[v]) {
                    *d_changed = true;
                    d_distance[v] = new_distance;
                }
            }
        }
    }
}

void bellman_ford_withThread(int *weights, int *distance, int start, int blkdim, bool *has_negative_cycle) {

    int iter_num = 0;
    int *d_weights, *d_distance;
    bool *d_changed, h_changed;

    // initializing the distance array
    for (int i = 0; i < VERTICES; i++) {
        distance[i] = INF;
    }
    distance[start] = 0;

    // Allocate GPU memory for d_weights, d_distance, d_changed
    hipMalloc(&d_weights, sizeof(int) * VERTICES * VERTICES);
    hipMalloc(&d_distance, sizeof(int) * VERTICES);
    hipMalloc(&d_changed, sizeof(bool));

    //Transfer the data from host to GPU.
    hipMemcpy(d_weights, weights, sizeof(int) * VERTICES * VERTICES, hipMemcpyHostToDevice);
    hipMemcpy(d_distance, distance, sizeof(int) * VERTICES, hipMemcpyHostToDevice);

    for (;;) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_withThread_kernel<<<1, blkdim>>>(d_weights, d_distance, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if (iter_num >= VERTICES - 1) {
            *has_negative_cycle = true;
            break;
        }
        if (!h_changed) {
            break;
        }
    }
    if (!*has_negative_cycle) {
        // Copy the shortest path distances back to the host memory
        hipMemcpy(distance, d_distance, sizeof(int) * VERTICES, hipMemcpyDeviceToHost);
    }
    
    // Free up the GPU memory.
    hipFree(d_weights);
    hipFree(d_distance);
    hipFree(d_changed);
}

// withBlocksThreads =========================================================================================
__global__ void bellman_ford_kernel(int *d_weights, int *d_distance, bool *d_changed) {
    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    if (global_tid < VERTICES){
        for (int u = 0; u < VERTICES; u++) {
            for (int v = global_tid; v < VERTICES; v += elementSkip) {
                int weight = d_weights[u * VERTICES + v];
                if (weight < INF) {
                    int new_distance = d_distance[u] + weight;
                    if (new_distance < d_distance[v]) {
                        *d_changed = true;
                        d_distance[v] = new_distance;
                    }
                }
            }
        }
    }
}

void bellman_ford(int *weights, int *distance, int start, int blkdim, bool *has_negative_cycle) {
    dim3 blocks((VERTICES + blkdim - 1) / blkdim);
    dim3 threads(blkdim);

    int iter_num = 0;
    int *d_weights, *d_distance;
    bool *d_changed, h_changed;

    // initializing the distance array
    for (int i = 0; i < VERTICES; i++) {
        distance[i] = INF;
    }
    distance[start] = 0;

    // Allocate GPU memory for d_weights, d_distance, d_changed
    hipMalloc(&d_weights, sizeof(int) * VERTICES * VERTICES);
    hipMalloc(&d_distance, sizeof(int) * VERTICES);
    hipMalloc(&d_changed, sizeof(bool));

    //Transfer the data from host to GPU.
    hipMemcpy(d_weights, weights, sizeof(int) * VERTICES * VERTICES, hipMemcpyHostToDevice);
    hipMemcpy(d_distance, distance, sizeof(int) * VERTICES, hipMemcpyHostToDevice);

    for (;;) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_kernel<<<blocks, threads>>>(d_weights, d_distance, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if (iter_num >= VERTICES - 1) {
            *has_negative_cycle = true;
            break;
        }
        if (!h_changed) {
            break;
        }
    }
    if (!*has_negative_cycle) {
        // Copy the shortest path distances back to the host memory
        hipMemcpy(distance, d_distance, sizeof(int) * VERTICES, hipMemcpyDeviceToHost);
    }
    
    // Free up the GPU memory.
    hipFree(d_weights);
    hipFree(d_distance);
    hipFree(d_changed);
}

// ===========================================================================================================

int main(int argc, char **argv) {
    // make sure we pass blockPerGrid and threadsPerBlock
    assert(argv[1] != NULL);
    int blkdim = atoi(argv[1]);

    int n_edges;

    // initializing distance array
    int* distance = (int*)malloc(VERTICES * sizeof(int));
    // reading the adjacency matrix
    int* weights = (int*)malloc(VERTICES * VERTICES * sizeof(int));
    read_file("data/USA-road-NY.csv", weights, &n_edges);

    bool has_negative_cycle = false;
    double tstart, tend;
            
    printf("CUDA Specifications ==================\n");

    // recored the execution time
    hipDeviceReset();
    tstart = gettime();
    bellman_ford_sequential(weights, distance, 0, &has_negative_cycle);
    hipDeviceSynchronize();
    tend = gettime();

    printf("Sequential Implementation\n");
    printf("(blocks, threads):\t(1, 1)\n");
    printf("Exection time:\t\t%.6f sec\n\n", tend-tstart);

    // recored the execution time
    hipDeviceReset();
    tstart = gettime();
    bellman_ford_withBlock(weights, distance, 0, &has_negative_cycle);
    hipDeviceSynchronize();
    tend = gettime();

    printf("Block Parallel Implementation\n");
    printf("(blocks, threads):\t(%d, 1)\n", VERTICES);
    printf("Exection time:\t\t%.6f sec\n\n", tend-tstart);

    // recored the execution time
    hipDeviceReset();
    tstart = gettime();
    bellman_ford_withThread(weights, distance, 0, blkdim, &has_negative_cycle);
    hipDeviceSynchronize();
    tend = gettime();

    printf("Thread Implementation\n");
    printf("(blocks, threads):\t(1, %d)\n", blkdim);
    printf("Exection time:\t\t%.6f sec\n\n", tend-tstart);

    // recored the execution time
    hipDeviceReset();
    tstart = gettime();
    bellman_ford(weights, distance, 0, blkdim, &has_negative_cycle);
    hipDeviceSynchronize();
    tend = gettime();

    printf("Thread/Block Implementation\n");
    printf("(blocks, threads):\t(%d, %d)\n", ((VERTICES+blkdim-1)/blkdim), blkdim);
    printf("Exection time:\t\t%.6f sec\n\n", tend-tstart);

    save_results(distance, has_negative_cycle);

    return 0;
}
