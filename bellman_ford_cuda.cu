
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <limits.h>
#include <assert.h>
#include <time.h>
// #include "hpc.h"

#define INF 999999
#define VERTICES 20000 //total vertices 264,346


double gettime(void){
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

void read_file(const char* filename, int* weights, int* n_edges) {
    // Initialize the matrix with INF and 0 for diagonals
    for (int i = 0; i < VERTICES; i++) {
        for (int j = 0; j < VERTICES; j++) {
            if (i != j) {
                weights[i * VERTICES + j] = INF;
            } else {
                weights[i * VERTICES + j] = 0;
            }
        }
    }

    // Open the CSV file
    FILE* file = fopen(filename, "r");

    // Read each line in the CSV file and update the matrix
    char line[256];
    n_edges = 0;
    while (fgets(line, sizeof(line), file)) {
        char* token;
        char* rest = line;
        int src_id, dest_id, distance;

        // Tokenize the line based on the comma delimiter
        token = strtok_r(rest, ",", &rest);
        src_id = atoi(token);

        token = strtok_r(rest, ",", &rest);
        dest_id = atoi(token);

        token = strtok_r(rest, ",", &rest);
        distance = atoi(token);

        // Update the matrix with the distance value
        if (src_id < VERTICES && dest_id < VERTICES) {
            n_edges++;
            weights[src_id * VERTICES + dest_id] = distance;
        }
    }
    fclose(file);
}

void save_results(int *distance, bool has_negative_cycle) {
    FILE *outputf = fopen("cuda_output.txt", "w");
    if (!has_negative_cycle) {
        for (int i = 0; i < VERTICES; i++) {
            if (distance[i] > INT_MAX)
                distance[i] = INT_MAX;
            fprintf(outputf, "%d\n", distance[i]);
        }
        fflush(outputf);
    } else {
        fprintf(outputf, "Negative cycle detected!\n");
    }
    fclose(outputf);
}

__global__ void bellman_ford_kernel(int *d_weights, int *d_distance, int n, bool *d_changed) {
    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    if (global_tid < n){
        for (int u = 0; u < n; u++) {
            for (int v = global_tid; v < n; v += elementSkip) {
                int weight = d_weights[u * n + v];
                if (weight < INF) {
                    int new_distance = d_distance[u] + weight;
                    if (new_distance < d_distance[v]) {
                        *d_changed = true;
                        d_distance[v] = new_distance;
                    }
                }
            }
        }
    }
}

void bellman_ford(int *weights, int *distance, int start, int n, int blocksPerGrid, int threadsPerBlock, bool *has_negative_cycle) {
    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);

    int iter_num = 0;
    int *d_weights, *d_distance;
    bool *d_changed, h_changed;

    // initializing the distance array
    for (int i = 0; i < n; i++) {
        distance[i] = INF;
    }
    distance[start] = 0;

    // Allocate GPU memory for d_weights, d_distance, d_changed
    hipMalloc(&d_weights, sizeof(int) * n * n);
    hipMalloc(&d_distance, sizeof(int) * n);
    hipMalloc(&d_changed, sizeof(bool));

    //Transfer the data from host to GPU.
    hipMemcpy(d_weights, weights, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_distance, distance, sizeof(int) * n, hipMemcpyHostToDevice);

    for (;;) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_kernel<<<blocks, threads>>>(d_weights, d_distance, n, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if (iter_num >= n - 1) {
            *has_negative_cycle = true;
            break;
        }
        if (!h_changed) {
            break;
        }
    }
    if (!*has_negative_cycle) {
        // Copy the shortest path distances back to the host memory
        hipMemcpy(distance, d_distance, sizeof(int) * n, hipMemcpyDeviceToHost);
    }
    
    // Free up the GPU memory.
    hipFree(d_weights);
    hipFree(d_distance);
    hipFree(d_changed);
}

int main(int argc, char **argv) {
    // make sure we pass blockPerGrid and threadsPerBlock
    assert(argv[1] != NULL && argv[2]!=NULL);
    int blocksPerGrid = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);

    int n_edges;
    
    // reading the adjacency matrix
    int* weights = (int*)malloc(VERTICES * VERTICES * sizeof(int));
    read_file("data/USA-road-NY.csv", weights, &n_edges);

    // initializing distance array
    int* distance = (int*)malloc(VERTICES * sizeof(int));

    bool has_negative_cycle = false;

    double tstart, tend;

    // recored the execution time
    hipDeviceReset();
    tstart = gettime();
    bellman_ford(weights, distance, 0, VERTICES, blocksPerGrid, threadsPerBlock, &has_negative_cycle);
    hipDeviceSynchronize();
    tend = gettime();

    printf("CUDA Specifications-------------\n");
    printf("blockPerGrid:\t\t%d\n", blocksPerGrid);
    printf("threadsPerBlock:\t%d\n", threadsPerBlock);
    printf("Exection time:\t\t%.6f sec\n\n", tend-tstart);

    save_results(distance, has_negative_cycle);

    return 0;
}
